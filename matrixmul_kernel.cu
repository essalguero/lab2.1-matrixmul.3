#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* Matrix multiplication: P = M * N.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"


////////////////////////////////////////////////////////////////////////////////
//! Matrix multiplication on the device: P = M * N
//! Mw is M's width and Nw is N's width
////////////////////////////////////////////////////////////////////////////////
    __global__ void
matrixMul( float* P, float* M, float* N, int Mw, int Nw)
{
    int bx = blockIdx.x;     int by = blockIdx.y;
    int tx = threadIdx.x;    int ty = threadIdx.y;
    __shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

    // ===================================================================
    // Code segment 1
    // Determine the update values for the tile indices in the loop
    // ===================================================================
	
	// Si el tamaño del bloque es menor que el tamaño de fila,
	// dividir el copiado de filas en bloques

	// Total de bloques en que se dividen filas y columnas
	int totalBloques = Mw / BLOCK_SIZE;

int sizeFila = gridDim.y * blockDim.y;
int fila = (blockDim.x * bx) + tx;
int columna = (blockDim.y * by) + ty;
int posicion = (fila * sizeFila) + columna;

int posicionFila = fila * sizeFila;
int posicionColumna = (blockDim.y * by) + ty;


	// Utilizado para depurar
	/*if (bx == 1 && by == 1) {
		printf("Numero de bloques por fila: %d, Numero Threads por bloque: %d, Threads en X: %d, Threads en Y: %d , fila = %d, columna: %d, posicion: %d\n", totalBloques, threadsPorBloque, blockDim.x, blockDim.y, posicionFila, columna, posicion);

	}*/


    // ===================================================================
    // Code segment 2
    // Do matrix-matrix multiplication inside a tile
    // ===================================================================
    int contadorBloque;
    float pSub = 0;

    // Cada fila y columna esta dividida en un numero de bloques de tamaño BLOCK_SIZE
    for (contadorBloque = 0; contadorBloque < totalBloques; ++contadorBloque) {

        // Load a tile from M and N into the shared memory arrays

	// Calcular la posicion de acceso a M (filas a desplazarse)
	int posicionM = posicionFila + (contadorBloque * BLOCK_SIZE) + ty;

	// Utilizado para depurar
	/*if ((bx == 0 && by == 0)) {
		printf("tx: %d, ty: %d, contadorBloque: %d, posicionM: %d\n", tx, ty, contadorBloque, posicionM);
	}*/

	// Calcular la posicion de acceso a N (columnas a desplazarse)
	int posicionN = posicionColumna + (tx * sizeFila) + (sizeFila * contadorBloque * BLOCK_SIZE);

	// Utilizado para depurar
	/*if ((bx == 0 && by == 0)) {
		printf("tx: %d, ty: %d, contadorBloque: %d, posicionN: %d\n", tx, ty, contadorBloque, posicionN);
	}*/

	Ms[tx][ty] = M[posicionM];
	Ns[tx][ty] = M[posicionN];

        // Synchronize the threads
	__syncthreads();
	
        // Multiply the two tiles together, each thread accumulating
        // the partial sum of a single dot product.
        for (int i = 0; i < BLOCK_SIZE; i++) { // bucle dado
		// En la iteracion se obtiene un valor parcial de la casilla
		pSub += Ms[tx][i] * Ns[i][ty];
        }

        // Synchronize again.
	__syncthreads();

	// Utilizado para depurar
	/*if ((bx == 1 && by == 1)) {
		printf("tx: %d, ty: %d, Suma: %f\n", tx, ty, pSub);
	}*/

    }

    // ===================================================================
    // Code segment 3
    // Store the data back to global memory
    // ===================================================================

	P[posicion] = pSub;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
